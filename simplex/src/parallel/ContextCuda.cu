//////////////////////////////////////////////////////////////////////////
// Auxiliary Function CUDA
// Copyright (c) (2018-), Bo Zhu
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "ContextCuda.h"

////Auxiliary variables
hipblasHandle_t cublas_handle=0;
hipsparseHandle_t cusparse_handle=0;
hipsparseMatDescr_t cusparse_mat_descr=0;
hipsolverSpHandle_t cusolver_handle=0;
hipblasStatus_t cublas_status;
bool cu_context_initialized=false;

bool Is_Cuda_Context_Initialized()
{return cu_context_initialized;}

hipblasHandle_t Cublas_Handle()
{return cublas_handle;}

hipsparseHandle_t Cusparse_Handle()
{return cusparse_handle;}

hipsolverSpHandle_t Cusolver_Handle()
{return cusolver_handle;}

hipsparseMatDescr_t Cusparse_Mat_Descr()
{return cusparse_mat_descr;}

void Initialize_Cuda_Context()
{
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t=hipblasCreate(&cublas_handle);
    checkCudaErrors(hipblasStatus_t);
    hipsparseStatus_t cusparseStatus;
    cusparseStatus=hipsparseCreate(&cusparse_handle);
    checkCudaErrors(cusparseStatus);
    cusparseStatus=hipsparseCreateMatDescr(&cusparse_mat_descr);
    checkCudaErrors(cusparseStatus);
    hipsparseSetMatType(cusparse_mat_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(cusparse_mat_descr,HIPSPARSE_INDEX_BASE_ZERO);

	hipsolverStatus_t cusolverStatus;
	cusolverStatus=hipsolverSpCreate(&cusolver_handle);
	checkCudaErrors(cusolverStatus);

	cu_context_initialized=true;
}
//////////////////////////////////////////////////////////////////////////
// Auxiliary Function CUDA
// Copyright (c) (2018-), Bo Zhu, Mengdi Wang
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "ContextCuda.h"
#include "AuxFuncCuda.h"

namespace AuxFuncCuda
{
	
static double one_d=1.;
static double zero_d=0.;
static double neg_one_d=-1.;
static float one_f=1.;
static float zero_f=0.;
static float neg_one_f=-1.;
hipblasStatus_t cublas_status;

//////////////////////////////////////////////////////////////////////////
////Basic APIs

////Data holder calculation
DataHolder Copy_Data_Holder(const DataHolder& to, const DataHolder& from) {
	if (from == UNKNOWN) {
		std::cerr << "[Error] AuxFuncCuda::Copy_Data_Holder: from=UNKNOWN\n";
		assert(false);
	}
	if (to == UNKNOWN) return from;
	else return to;
}

////Memory allocation, communication between CPU and GPU

template<class T> T* Global_Malloc(int n, const DataHolder &side) {
	T* p = nullptr;
	if (side == UNKNOWN) { std::cerr << "[Error] AuxFuncCuda::Global_Malloc: unknown data holder\n"; }
	else if (side == HOST) { p = new T[n]; }
	else { checkCudaErrors(hipMalloc((void**)&p, n * sizeof(T))); }
	return p;
}

template<class T> T* Global_Free(T*& ptr, const DataHolder& side) {
	if (ptr == nullptr) return nullptr;
	if (side == UNKNOWN) { std::cerr << "[Error] AuxFuncCuda::Global_Malloc: unknown data holder\n"; }
	else if (side == HOST) { delete[] ptr; }
	else if (side == DEVICE) { checkCudaErrors(hipFree((void*)ptr)); }
	return nullptr;
}

template<class T> void Global_Copy_Array(T*& ptr_to, T*& ptr_from, int n, const DataHolder& to_side, const DataHolder& from_side) {
	size_t bytenum = sizeof(T) * n;
	hipMemcpyKind op_kind;
	if (from_side == HOST && to_side == HOST) { op_kind = hipMemcpyHostToHost; }
	else if (from_side == HOST && to_side == DEVICE) { op_kind = hipMemcpyHostToDevice; }
	else if (from_side == DEVICE && to_side == HOST) { op_kind = hipMemcpyDeviceToHost; }
	else if (from_side == DEVICE && to_side == DEVICE) { op_kind = hipMemcpyDeviceToDevice; }
	else { std::cerr << "[Error] AuxFuncCuda::Global_Copy_Array: unknown data holder\n"; }
	hipMemcpy(ptr_to, ptr_from, bytenum, op_kind);
}

template<class T> T* Global_Malloc_And_Copy_Array(T*& ptr_from, int n, const DataHolder& to_side, const DataHolder& from_side) {
	T* ptr_to = Global_Malloc<T>(n, to_side);
	Global_Copy_Array<T>(ptr_to, ptr_from, n, to_side, from_side);
	return ptr_to;
}
template<class T> void Global_Realloc_And_Copy_Array(T*& ptr_to, T*& ptr_from, int n, const DataHolder& to_side, const DataHolder& from_side) {
	Global_Free<T>(ptr_to, to_side);
	ptr_to = Global_Malloc_And_Copy_Array<T>(ptr_from, n, to_side, from_side);
}

//only 1 element
template<class T> void Global_Realloc_And_Copy(T*& ptr_to, T*& ptr_from, const DataHolder& to_side, const DataHolder& from_side)
{
	Global_Realloc_And_Copy_Array(ptr_to, ptr_from, 1, to_side, from_side);
}

template<class T> void Global_Memset(T* ptr, int v, int n, const DataHolder& side) {
	size_t bytenum = n * sizeof(T);
	if (side == HOST) { memset(ptr, v, bytenum); }
	else if (side == DEVICE) { hipMemset(ptr, v, bytenum); }
	else { std::cerr << "[Error] AuxFuncCuda::Global_Memset: unknown data type\n"; }
}

/*template<class T> void Malloc_On_Device(T*& ptr_dev,int n)
{ptr_dev = Global_Malloc<T>(n, DEVICE);}

template<class T> void Free_On_Device(T*& ptr_dev)
{Global_Free<T>(ptr_dev, DEVICE);}

template<class T> void Malloc_On_Host(T*& ptr_host, int n)
{ptr_host = Global_Malloc<T>(n, HOST);}

template<class T> void Free_On_Host(T*& ptr_host)
{Global_Free<T>(ptr_host, HOST);}

template<class T> void Copy_Array_Host_To_Device(T*& ptr_dev,T*& ptr_host,int n)
{Global_Copy_Array<T>(ptr_dev, ptr_host, n, DEVICE, HOST);}
template<class T> void Copy_Array_Device_To_Host(T*& ptr_host, T*& ptr_dev, int n)
{Global_Copy_Array<T>(ptr_host, ptr_dev, n, HOST, DEVICE);}
template<class T> void Copy_Array_Device_To_Device(T*& ptr_dev_to, T*& ptr_dev_from, int n)
{Global_Copy_Array<T>(ptr_dev_to, ptr_dev_from, n, DEVICE, DEVICE);}
template<class T> void Copy_Array_Host_To_Host(T*& ptr_host_to,T*& ptr_host_from,int n)
{Global_Copy_Array<T>(ptr_host_to, ptr_host_from, n, HOST, HOST);}


template<class T> void Malloc_And_Copy_Array_Host_To_Device(T*& ptr_dev, T*& ptr_host, int n)
{Global_Realloc_And_Copy_Array<T>(ptr_dev, ptr_host, n, DEVICE, HOST);}
template<class T> void Malloc_And_Copy_Array_Host_To_Host(T*& ptr_host_to,T*& ptr_host_from,int n)
{Global_Realloc_And_Copy_Array<T>(ptr_host_to, ptr_host_from, n, HOST, HOST);}
template<class T> void Malloc_And_Copy_Array_Device_To_Device(T*& ptr_dev_to,T*& ptr_dev_from,int n)
{Global_Realloc_And_Copy_Array<T>(ptr_dev_to, ptr_dev_from, n, DEVICE, DEVICE);}
template<class T> void Malloc_And_Copy_Array_Device_To_Host(T*& ptr_host_to, T*& ptr_dev_from, int n)
{Global_Realloc_And_Copy_Array<T>(ptr_host_to, ptr_dev_from, n, HOST, DEVICE);}



template<class T> void Malloc_And_Copy_Host_To_Device(T*& ptr_dev, T*& ptr_host)
{Global_Realloc_And_Copy<T>(ptr_dev, ptr_host, DEVICE, HOST);}
template<class T> void Malloc_And_Copy_Host_To_Host(T*& ptr_host_to,T*& ptr_host_from)
{Global_Realloc_And_Copy<T>(ptr_host_to, ptr_host_from, HOST, HOST);}
template<class T> void Malloc_And_Copy_Device_To_Device(T*& ptr_dev_to, T*& ptr_dev_from)
{Global_Realloc_And_Copy<T>(ptr_dev_to, ptr_dev_from, DEVICE, DEVICE);}

template<class T> void Memset_On_Device(T* x, int v, int n)
{Global_Memset<T>(x, v, n, DEVICE);}*/

#define Inst_Helper(T) \
template T* Global_Malloc<T>(int n, const DataHolder &side); \
template T* Global_Free<T>(T*& ptr, const DataHolder &side); \
template void Global_Copy_Array<T>(T*& ptr_to, T*& ptr_from, int n, const DataHolder &to_side, const DataHolder &from_side); \
template T* Global_Malloc_And_Copy_Array<T>(T*& ptr_from, int n, const DataHolder& to_side, const DataHolder& from_side); \
template void Global_Realloc_And_Copy_Array<T>(T*& ptr_to, T*& ptr_from, int n, const DataHolder& to_side, const DataHolder& from_side) ;\
template void Global_Realloc_And_Copy<T>(T*& ptr_to, T*& ptr_from, const DataHolder& to_side, const DataHolder& from_side);\
template void Global_Memset<T>(T* ptr, int v, int n, const DataHolder& side);/*\
template void Malloc_On_Device<T>(T*&,int); \
template void Free_On_Device<T>(T*&); \
template void Malloc_On_Host<T>(T*&,int); \
template void Free_On_Host<T>(T*&); \
template void Copy_Array_Host_To_Device<T>(T*&,T*&,int); \
template void Copy_Array_Device_To_Host<T>(T*&,T*&,int); \
template void Copy_Array_Device_To_Device<T>(T*&,T*&,int); \
template void Copy_Array_Host_To_Host<T>(T*&,T*&,int); \
template void Malloc_And_Copy_Array_Host_To_Device<T>(T*&,T*&,int); \
template void Malloc_And_Copy_Array_Host_To_Host<T>(T*&,T*&,int); \
template void Malloc_And_Copy_Array_Device_To_Device<T>(T*&,T*&,int); \
template void Malloc_And_Copy_Array_Device_To_Host<T>(T*& ptr_host_to,T*& ptr_dev_from,int n);\
template void Malloc_And_Copy_Host_To_Device<T>(T*&,T*&); \
template void Malloc_And_Copy_Host_To_Host<T>(T*&,T*&); \
template void Malloc_And_Copy_Device_To_Device<T>(T*&,T*&); \
template void Memset_On_Device<T>(T*,int,int);*/
Inst_Helper(int);
Inst_Helper(float);
Inst_Helper(double);
#undef Inst_Helper

template<class T>
hipsparseDnVecDescr_t Create_DnVecDescr_t(T* x, int size) {//x must be on device
	hipsparseDnVecDescr_t vec_t = nullptr;
	hipsparseStatus_t stat = hipsparseCreateDnVec(&vec_t, size, x, Cuda_Real_Type<T>());
	return vec_t;
}

////CUDA11 TOFIX [JY]: csrmv
////Linear algebra operations on device
template<class T>//T={double, float}
void Csrmv(SparseMatrixCuda<T>* A, T* x, T* y, T* alpha, T* beta)//y=alpha*A*x+beta*y
{
	//A,x,y must be on device
	//This function will internally allocate and erase temporary space dBuffer
	hipsparseHandle_t handle = nullptr; hipsparseCreate(&handle);
	hipsparseSpMatDescr_t A_desc = A->Get_SpMatCescr_t();
	hipsparseDnVecDescr_t x_desc = Create_DnVecDescr_t(x, A->n);
	hipsparseDnVecDescr_t y_desc = Create_DnVecDescr_t(y, A->m);
	size_t buffersize; void* dBuffer = nullptr;
	hipsparseSpMV_bufferSize(
		handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		alpha, A_desc, x_desc, beta, y_desc, A->data_type,
		HIPSPARSE_MV_ALG_DEFAULT, &buffersize);
	hipMalloc(&dBuffer, buffersize);
	hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		alpha, A_desc, x_desc, beta, y_desc, A->data_type,
		HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
	hipsparseDestroy(handle);
	Global_Free(dBuffer, DEVICE);
}

void Mv(SparseMatrixCuda<double>* A,double* x,double* y)//y=alpha*a
{Csrmv(A,x,y,&one_d,&zero_d);}
void Mv(SparseMatrixCuda<float>* A,float* x,float* y)
{Csrmv(A,x,y,&one_f,&zero_f);}

////CUDA11 TOFIX [JY]: csrgemm
////example: https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spgemm/spgemm_example.c
template<class T>//T={double, float}
void SpGEMM(SparseMatrixCuda<T>* A, SparseMatrixCuda<T>* B, SparseMatrixCuda<T>* C) {//C=A*B
	assert(C != nullptr);
	//A,B must on device
	//This function will Re-allocate C on device
	//and internally allocate and erase temporary space dBuffer1, dBuffer2
	static hipsparseOperation_t no_op = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	static T alpha = 1, beta = 0;
	hipsparseHandle_t handle = nullptr; hipsparseCreate(&handle);
	hipsparseSpGEMMDescr_t spgemmDesc; hipsparseSpGEMM_createDescr(&spgemmDesc);
	hipsparseSpMatDescr_t A_desc = A->Get_SpMatCescr_t(), B_desc = B->Get_SpMatCescr_t(), C_desc = nullptr;
	hipsparseCreateCsr(&C_desc, A->rows(), B->cols(), 0, nullptr, nullptr, nullptr,
		HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, A->data_type);
	
	size_t bufferSize1 = 0, bufferSize2 = 0;
	void* dBuffer1 = nullptr, * dBuffer2 = nullptr;
	//calculate size of buffer 1
	hipsparseSpGEMM_workEstimation(handle, no_op, no_op, &alpha, A_desc, B_desc, &beta, C_desc,
		A->data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize1, nullptr);
	hipMalloc((void**)&dBuffer1, bufferSize1);
	//estimate with buffer 1
	hipsparseSpGEMM_workEstimation(handle, no_op, no_op, &alpha, A_desc, B_desc, &beta, C_desc,
		A->data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize1, dBuffer1);
	//calculate size of buffer 2
	hipsparseSpGEMM_compute(handle, no_op, no_op, &alpha, A_desc, B_desc, &beta, C_desc,
		A->data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize2, nullptr);
	hipMalloc((void**)&dBuffer2, bufferSize2);
	//compute with buffer 2
	hipsparseSpGEMM_compute(handle, no_op, no_op, &alpha, A_desc, B_desc, &beta, C_desc,
		A->data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize2, dBuffer2);
	int64_t C_num_rows, C_num_cols, C_nnz;
	hipsparseSpMatGetSize(C_desc, &C_num_rows, &C_num_cols, &C_nnz);
	C->resize(C_num_rows, C_num_cols, DEVICE);
	C->resizeNonZeros(C_nnz);
	hipsparseCsrSetPointers(C_desc, C->ptr, C->col, C->val);
	hipsparseSpGEMM_copy(handle, no_op, no_op, &alpha, A_desc, B_desc, &beta, C_desc,
		A->data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

	hipsparseSpGEMM_destroyDescr(spgemmDesc);
	Global_Free(dBuffer1, DEVICE);
	Global_Free(dBuffer2, DEVICE);
}

void Mm(SparseMatrixCuda<double>* A_dev, SparseMatrixCuda<double>* B_dev, SparseMatrixCuda<double>* C_dev)
{
	SpGEMM(A_dev, B_dev, C_dev);
}
void Mm(SparseMatrixCuda<float>* A_dev, SparseMatrixCuda<float>* B_dev, SparseMatrixCuda<float>* C_dev)
{
	SpGEMM(A_dev, B_dev, C_dev);
}

void Axpy(double* alpha,double* x,double* y,int n){hipblasDaxpy(Cublas_Handle(),n,alpha,x,1,y,1);}
void Axpy(float* alpha,float* x,float* y,int n){hipblasSaxpy(Cublas_Handle(),n,alpha,x,1,y,1);}

void Copy(double* to,double* from,int n){hipblasDcopy(Cublas_Handle(),n,from,1,to,1);}
void Copy(float* to,float* from,int n){hipblasScopy(Cublas_Handle(),n,from,1,to,1);}

double Dot(double* x,double* y,int n){double x_dot_y=0.;cublas_status=hipblasDdot(Cublas_Handle(),n,x,1,y,1,&x_dot_y);return x_dot_y;}
float Dot(float* x,float* y,int n){float x_dot_y=0.;cublas_status=hipblasSdot(Cublas_Handle(),n,x,1,y,1,&x_dot_y);return x_dot_y;}

void Scale(double* alpha,double* x,int n){cublas_status=hipblasDscal(Cublas_Handle(),n,alpha,x,1);}
void Scale(float* alpha,float* x,int n){cublas_status=hipblasSscal(Cublas_Handle(),n,alpha,x,1);}
};
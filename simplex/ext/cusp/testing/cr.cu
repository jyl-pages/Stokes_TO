#include "hip/hip_runtime.h"
#include <unittest/unittest.h>

#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>

#include <cusp/gallery/poisson.h>
#include <cusp/krylov/cr.h>

template <class LinearOperator, class VectorType1, class VectorType2, class Monitor, class Preconditioner>
void cr(my_system& system, const LinearOperator& A, VectorType1& x, const VectorType2& b, Monitor& monitor, Preconditioner& M)
{
    system.validate_dispatch();
    return;
}

void TestConjugateResidualDispatch()
{
    // initialize testing variables
    cusp::csr_matrix<int, float, cusp::device_memory> A;
    cusp::gallery::poisson5pt(A, 10, 10);
    cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0.0f);
    cusp::monitor<float> monitor(x, 20, 1e-4);
    cusp::identity_operator<float,cusp::device_memory> M(A.num_rows, A.num_cols);

    my_system sys(0);

    // call with explicit dispatching
    cusp::krylov::cr(sys, A, x, x, monitor, M);

    // check if dispatch policy was used
    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestConjugateResidualDispatch);

template <class MemorySpace>
void TestConjugateResidual(void)
{
    cusp::csr_matrix<int, float, MemorySpace> A;

    cusp::gallery::poisson5pt(A, 10, 10);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 0.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows, 1.0f);

    cusp::monitor<float> monitor(b, 20, 1e-4);

    cusp::krylov::cr(A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(cusp::blas::nrm2(residual) < 1e-4 * cusp::blas::nrm2(b), true);
}
DECLARE_HOST_DEVICE_UNITTEST(TestConjugateResidual);


template <class MemorySpace>
void TestConjugateResidualZeroResidual(void)
{
    cusp::array2d<float, MemorySpace> M(2,2);
    M(0,0) = 8;
    M(0,1) = 0;
    M(1,0) = 0;
    M(1,1) = 4;

    cusp::csr_matrix<int, float, MemorySpace> A(M);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 1.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows);

    cusp::multiply(A, x, b);

    cusp::monitor<float> monitor(b, 20, 0.0f);

    cusp::krylov::cr(A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(monitor.converged(),        true);
    ASSERT_EQUAL(monitor.iteration_count(),     0);
    ASSERT_EQUAL(cusp::blas::nrm2(residual), 0.0f);
}
DECLARE_HOST_DEVICE_UNITTEST(TestConjugateResidualZeroResidual);


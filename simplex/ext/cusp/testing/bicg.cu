#include "hip/hip_runtime.h"
#include <unittest/unittest.h>

#include <cusp/csr_matrix.h>
#include <cusp/linear_operator.h>
#include <cusp/monitor.h>
#include <cusp/multiply.h>

#include <cusp/gallery/poisson.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/bicgstab.h>

template <class LinearOperator,
          class VectorType1,
          class VectorType2,
          class Monitor,
          class Preconditioner>
void bicg(my_system& system,
          const LinearOperator& A,
          const LinearOperator& At,
                VectorType1& x,
          const VectorType2& b,
                Monitor& monitor,
                Preconditioner& M,
                Preconditioner& Mt)
{
    system.validate_dispatch();
    return;
}

void TestBiConjugateGradientDispatch()
{
    // initialize testing variables
    cusp::csr_matrix<int, float, cusp::device_memory> A;
    cusp::gallery::poisson5pt(A, 10, 10);
    cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0.0f);
    cusp::monitor<float> monitor(x, 20, 1e-4);
    cusp::identity_operator<float,cusp::device_memory> M(A.num_rows, A.num_cols);

    my_system sys(0);

    // call with explicit dispatching
    cusp::krylov::bicg(sys, A, A, x, x, monitor, M, M);

    // check if dispatch policy was used
    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestBiConjugateGradientDispatch);

template <class MemorySpace>
void TestBiConjugateGradient(void)
{
    cusp::csr_matrix<int, float, MemorySpace> A;

    cusp::gallery::poisson5pt(A, 10, 10);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 0.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows, 1.0f);

    cusp::monitor<float> monitor(b, 20, 1e-4);

    cusp::krylov::bicg(A, A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(cusp::blas::nrm2(residual) < 1e-4 * cusp::blas::nrm2(b), true);
}
DECLARE_HOST_DEVICE_UNITTEST(TestBiConjugateGradient)

template <class MemorySpace>
void TestBiConjugateGradientZeroResidual(void)
{
    cusp::array2d<float, MemorySpace> M(2,2);
    M(0,0) = 8;
    M(0,1) = 0;
    M(1,0) = 0;
    M(1,1) = 4;

    cusp::csr_matrix<int, float, MemorySpace> A(M);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 1.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows);

    cusp::multiply(A, x, b);

    cusp::monitor<float> monitor(b, 20, 0.0f);

    cusp::krylov::bicg(A, A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(monitor.converged(),        true);
    ASSERT_EQUAL(monitor.iteration_count(),     0);
    ASSERT_EQUAL(cusp::blas::nrm2(residual), 0.0f);
}
DECLARE_HOST_DEVICE_UNITTEST(TestBiConjugateGradientZeroResidual)

template <class LinearOperator, class VectorType1, class VectorType2, class Monitor, class Preconditioner>
void bicgstab(my_system& system, const LinearOperator& A, VectorType1& x, const VectorType2& b, Monitor& monitor, Preconditioner& M)
{
    system.validate_dispatch();
    return;
}

void TestBiConjugateGradientStabilizedDispatch()
{
    // initialize testing variables
    cusp::csr_matrix<int, float, cusp::device_memory> A;
    cusp::gallery::poisson5pt(A, 10, 10);
    cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0.0f);
    cusp::monitor<float> monitor(x, 20, 1e-4);
    cusp::identity_operator<float,cusp::device_memory> M(A.num_rows, A.num_cols);

    my_system sys(0);

    // call with explicit dispatching
    cusp::krylov::bicgstab(sys, A, x, x, monitor, M);

    // check if dispatch policy was used
    ASSERT_EQUAL(true, sys.is_valid());
}
DECLARE_UNITTEST(TestBiConjugateGradientStabilizedDispatch);

template <class MemorySpace>
void TestBiConjugateGradientStabilized(void)
{
    cusp::csr_matrix<int, float, MemorySpace> A;

    cusp::gallery::poisson5pt(A, 10, 10);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 0.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows, 1.0f);

    cusp::monitor<float> monitor(b, 20, 1e-4);

    cusp::krylov::bicgstab(A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(cusp::blas::nrm2(residual) < 1e-4 * cusp::blas::nrm2(b), true);
}
DECLARE_HOST_DEVICE_UNITTEST(TestBiConjugateGradientStabilized)

template <class MemorySpace>
void TestBiConjugateGradientStabilizedZeroResidual(void)
{
    cusp::array2d<float, MemorySpace> M(2,2);
    M(0,0) = 8;
    M(0,1) = 0;
    M(1,0) = 0;
    M(1,1) = 4;

    cusp::csr_matrix<int, float, MemorySpace> A(M);

    cusp::array1d<float, MemorySpace> x(A.num_rows, 1.0f);
    cusp::array1d<float, MemorySpace> b(A.num_rows);

    cusp::multiply(A, x, b);

    cusp::monitor<float> monitor(b, 20, 0.0f);

    cusp::krylov::bicgstab(A, x, b, monitor);

    // check residual norm
    cusp::array1d<float, MemorySpace> residual(A.num_rows, 0.0f);
    cusp::multiply(A, x, residual);
    cusp::blas::axpby(residual, b, residual, -1.0f, 1.0f);

    ASSERT_EQUAL(monitor.converged(),        true);
    ASSERT_EQUAL(monitor.iteration_count(),     0);
    ASSERT_EQUAL(cusp::blas::nrm2(residual), 0.0f);
}
DECLARE_HOST_DEVICE_UNITTEST(TestBiConjugateGradientStabilizedZeroResidual)

